#include "Convergence_GPU.hpp"
#include "kernel_GPU.cuh"

#include "hip/hip_runtime.h"

inline bool CUDA_MALLOC( void ** devPtr, size_t size ) {
	hipError_t cudaStatus;
	cudaStatus = hipMalloc( devPtr, size );
	if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to allocate buffer\n");
		return false;
	}
	return true;
}

inline bool CUDA_MEMCPY( void * dst, const void * src, size_t count, enum hipMemcpyKind kind ) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy( dst, src, count, kind );
	if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to copy buffer\n");
		return false;
	}
	return true;
}

Convergence_GPU::Convergence_GPU() : Convergence("GPU_double")
{

}


Convergence_GPU::Convergence_GPU(ColorMap* _colors, int _max_iters) : Convergence("GPU_double")
{
    colors    = _colors;
    max_iters = _max_iters;

    hostTab = nullptr;
    deviceTab = nullptr;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to setup cuda device\n");
        exit(0);
	}
}


Convergence_GPU::~Convergence_GPU( ){
     hipError_t cudaStatus = hipDeviceReset();
     free(hostTab);
     free(deviceTab);
}

void Convergence_GPU::updateImage(const long double _zoom, const long double _offsetX, const long double _offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, sf::Image& image)
{
    int nb_point = IMAGE_WIDTH*IMAGE_HEIGHT;
    dim3 grid(80,50,1); //nbr bloc
    dim3 block(16,16,1); //nbr threads

    if(hostTab == nullptr)
        hostTab = new uint32_t[nb_point];

    if(deviceTab == nullptr)
        CUDA_MALLOC((void**)&deviceTab, nb_point * sizeof(uint32_t));

    double offsetX = _offsetX;
    double offsetY = _offsetX;
    double zoom    = _zoom;

    kernel_updateImage_GPU<<<grid, block>>>(zoom, offsetX, offsetY, IMAGE_WIDTH, IMAGE_HEIGHT, deviceTab, max_iters);

    CUDA_MEMCPY(hostTab, deviceTab, nb_point*sizeof(uint32_t), hipMemcpyDeviceToHost);


    for(int y = 0; y < IMAGE_HEIGHT; y++)
    {
        for(int x = 0; x < IMAGE_WIDTH; x++)
        {
            image.setPixel(x, y, colors->getColor(hostTab[x+y*IMAGE_WIDTH]));
        }
    }
}