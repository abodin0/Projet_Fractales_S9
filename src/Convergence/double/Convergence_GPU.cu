#include "Convergence_GPU.hpp"
#include "Calcul_GPU.cuh"

#include "hip/hip_runtime.h"

inline bool CUDA_MALLOC( void ** devPtr, size_t size ) {
	hipError_t cudaStatus;
	cudaStatus = hipMalloc( devPtr, size );
	if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to allocate buffer\n");
		return false;
	}
	return true;
}

inline bool CUDA_MEMCPY( void * dst, const void * src, size_t count, enum hipMemcpyKind kind ) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy( dst, src, count, kind );
	if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to copy buffer\n");
		return false;
	}
	return true;
}

Convergence_GPU::Convergence_GPU() : Convergence("GPU")
{

}


Convergence_GPU::Convergence_GPU(ColorMap* _colors, int _max_iters) : Convergence("GPU")
{
    colors    = _colors;
    max_iters = _max_iters;

    hostTab = nullptr;
    deviceTab = nullptr;
}


Convergence_GPU::~Convergence_GPU( ){
     hipError_t cudaStatus = hipDeviceReset();
     free(hostTab);
     free(deviceTab);
}

void Convergence_GPU::updateImage(const long double _zoom, const long double _offsetX, const long double _offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, sf::Image& image)
{
    int nb_point = IMAGE_WIDTH*IMAGE_HEIGHT;
    int nthreads = 1024;
    int nblocks = ( nb_point + ( nthreads - 1 ) ) / nthreads;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to setup cuda device\n");
	}

    if(hostTab == nullptr)
        hostTab = new uint32_t[nb_point];

    for(int i = 0 ; i<nb_point; i++){
        hostTab[i] = i%256;
    }

    if(deviceTab == nullptr){
        CUDA_MALLOC((void**)&deviceTab, nb_point * sizeof(uint32_t));
        //CUDA_MEMCPY(deviceTab, hostTab, nb_point*sizeof(uint32_t), hipMemcpyHostToDevice);
    }

    //kernel_updateImage_GPU<<<nblocks, nthreads>>>(_zoom, _offsetX, _offsetY, IMAGE_WIDTH, IMAGE_HEIGHT, deviceTab, max_iters);

    CUDA_MEMCPY(hostTab, deviceTab, nb_point*sizeof(uint32_t), hipMemcpyDeviceToHost);


    for(int y = 0; y < IMAGE_HEIGHT; y++)
    {
        for(int x = 0; x < IMAGE_WIDTH; x++)
        {
            image.setPixel(x, y, colors->getColor(hostTab[x+y*IMAGE_WIDTH]));
        }
    }
    printf("value (CPU) = %x \n", hostTab[50+200*IMAGE_WIDTH]);
}