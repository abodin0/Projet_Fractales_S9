#include "hip/hip_runtime.h"
#include "Convergence_dp_x86.cuh"

Convergence_dp_x86::Convergence_dp_x86() : Convergence("DP")
{

}


Convergence_dp_x86::Convergence_dp_x86(ColorMap* _colors, int _max_iters) : Convergence("DP")
{
    colors    = _colors;
    max_iters = _max_iters;
}


Convergence_dp_x86::~Convergence_dp_x86( ){

}


__global__ void Convergence_dp_x86::kernel_updateImage(const long double _zoom, const long double _offsetX, const long double _offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, uint32_t * deviceTab)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int nb_point = IMAGE_WIDTH * IMAGE_HEIGHT;
    int x, y = 0;

    double offsetX = _offsetX;
    double offsetY = _offsetX;
    double zoom    = _zoom;

    if(i<nb_point)
    {
      x = i%IMAGE_WIDTH;
      y = (i-x)/IMAGE_WIDTH;

      double startImag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
      double startReal = offsetX - IMAGE_WIDTH  / 2.0f * zoom;
      int value    = max_iters - 1;
      double zReal = startReal;
      double zImag = startImag;

      for (unsigned int counter = 0; counter < max_iters; counter++) 
      {
          double r2 = zReal * zReal;
          double i2 = zImag * zImag;
          zImag = 2.0f * zReal * zImag + startImag;
          zReal = r2 - i2 + startReal;
          if ( (r2 + i2) > 4.0f) {
              value = counter;
              break;
          }
      }

      deviceTab[x+y*IMAGE_WIDTH] = value;
      //image.setPixel(x, y, colors->getColor(value));
      startReal += zoom;
    }
}

void Convergence_dp_x86::updateImage(int nblocks, int nthreads, const long double _zoom, const long double _offsetX, const long double _offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, uint32_t * deviceTab)
{
  kernel_updateImage<<<nblocks, nthreads>>>(_zoom, _offsetX, _offsetY, IMAGE_WIDTH, IMAGE_HEIGHT, deviceTab);
}
