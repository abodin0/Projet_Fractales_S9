#include "hip/hip_runtime.h"
#include "kernel_GPU_mme.cuh"
#include "hip/hip_runtime.h"

__global__ void kernel_updateImage_GPU_mme(const double zoom, const double offsetX, const double offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, uint32_t * deviceTab, int max_iters)
{
    int blockID = blockIdx.x + (blockIdx.y * gridDim.x);
    int threadID = blockID * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    int y = threadID/IMAGE_WIDTH;
    int x = threadID%IMAGE_WIDTH;

    double startReal = offsetX - IMAGE_WIDTH  / 2.0f * zoom + (x * zoom);
    double startImag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    
    int value    = max_iters - 1;

    double zReal = startReal;
    double zImag = startImag;

    for (unsigned int counter = 0; counter < max_iters; counter++) 
    {
        zReal = abs(zReal);
        zImag = -zImag;

        double r2 = zReal * zReal;
        double r3 = zReal * zReal * zReal;
        double i2 = zImag * zImag;
        double i3 = zImag * zImag * zImag;
        
        zImag = 3.0f * r2 * zImag - i3 + startImag;
        zReal = r3 - 3.0f * zReal * i2 + startReal;

        if ( (r2 + i2) > 4.0f) {
            value = counter;
            break;
        }
    }
    deviceTab[y*IMAGE_WIDTH+x] = value;
}