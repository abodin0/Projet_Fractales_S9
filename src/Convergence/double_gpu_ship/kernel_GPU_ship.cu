#include "hip/hip_runtime.h"
#include "kernel_GPU_ship.cuh"
#include "hip/hip_runtime.h"

__global__ void kernel_updateImage_GPU_ship(const double zoom, const double offsetX, const double offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, uint32_t * deviceTab, int max_iters)
{
    int blockID = blockIdx.x + (blockIdx.y * gridDim.x);
    int threadID = blockID * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    int y = threadID/IMAGE_WIDTH;
    int x = threadID%IMAGE_WIDTH;

    double startImag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    double startReal = offsetX - IMAGE_WIDTH  / 2.0f * zoom + (x * zoom);

    int value    = max_iters - 1;
    double zReal = startReal;
    double zImag = startImag;

    for (unsigned int counter = 0; counter < max_iters; counter++) 
    {
        zImag = abs(zImag);
        zReal = abs(zReal);

        double r2 = zReal * zReal;
        double i2 = zImag * zImag;
        
        zImag = 2.0f * zReal * zImag + startImag;
        zReal = r2 - i2 + startReal;

        if ( (r2 + i2) > 4.0f) {
            value = counter;
            break;
        }
    }
    deviceTab[y*IMAGE_WIDTH+x] = value;
}