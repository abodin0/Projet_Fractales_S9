#include "Convergence_GPU_ship.hpp"
#include "kernel_GPU_ship.cuh"

#include "hip/hip_runtime.h"

inline bool CUDA_MALLOC( void ** devPtr, size_t size ) {
	hipError_t cudaStatus;
	cudaStatus = hipMalloc( devPtr, size );
	if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to allocate buffer\n");
		return false;
	}
	return true;
}

inline bool CUDA_MEMCPY( void * dst, const void * src, size_t count, enum hipMemcpyKind kind ) {
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy( dst, src, count, kind );
	if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to copy buffer\n");
		return false;
	}
	return true;
}

Convergence_GPU_ship::Convergence_GPU_ship() : Convergence("GPU_double_ship")
{

}


Convergence_GPU_ship::Convergence_GPU_ship(ColorMap* _colors, int _max_iters) : Convergence("GPU_double_ship")
{
    colors    = _colors;
    max_iters = _max_iters;

    hostTab = nullptr;
    deviceTab = nullptr;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if ( cudaStatus != hipSuccess ) {
		printf( "error: unable to setup cuda device\n");
        exit(0);
	}
}


Convergence_GPU_ship::~Convergence_GPU_ship( ){
     hipError_t cudaStatus = hipDeviceReset();
     free(hostTab);
     free(deviceTab);
}

void Convergence_GPU_ship::updateImage(const long double _zoom, const long double _offsetX, const long double _offsetY, const int IMAGE_WIDTH, const int IMAGE_HEIGHT, sf::Image& image)
{
    int nb_point = IMAGE_WIDTH*IMAGE_HEIGHT;
    dim3 grid(80,50,1); //nbr bloc
    dim3 block(16,16,1); //nbr threads

    if(hostTab == nullptr)
        hostTab = new uint32_t[nb_point];

    if(deviceTab == nullptr)
        CUDA_MALLOC((void**)&deviceTab, nb_point * sizeof(uint32_t));

    double offsetX = _offsetX;
    double offsetY = _offsetX;
    double zoom    = _zoom;

    kernel_updateImage_GPU_ship<<<grid, block>>>(zoom, offsetX, offsetY, IMAGE_WIDTH, IMAGE_HEIGHT, deviceTab, max_iters);

    CUDA_MEMCPY(hostTab, deviceTab, nb_point*sizeof(uint32_t), hipMemcpyDeviceToHost);


    for(int y = 0; y < IMAGE_HEIGHT; y++)
    {
        for(int x = 0; x < IMAGE_WIDTH; x++)
        {
            image.setPixel(x, y, colors->getColor(hostTab[x+y*IMAGE_WIDTH]));
        }
    }
}